//#include "kernel.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"


#define N 32768

__host__
bool checkArr(int *arr, int size)
{
	for (int i = 0; i < size-1; ++i)
	{
		if (arr[i] > arr[i + 1])
		{
			printf("Array index: %d, with value: %d\nIs greater than index: %d, with value: %d\n", i, arr[i], i + 1, arr[i + 1]);
		}
	}
	return true;
}

__host__
void printArr(int *arr, int size)
{
	for (int i = 0; i < size - 1; ++i)
	{
		printf("%d, ", arr[i]);
	}
	printf("%d \n\n", arr[size - 1]);
}

__host__
void createRandArr(int *arr, int size, int maxVal)
{
	for (int i = 0; i < size; ++i)
	{
		int rnd = (rand() / (float)(RAND_MAX)) * maxVal;
		arr[i] = rnd;
	}
}

__host__
int oddeven(int *arr, int size, int oddeven)
{
	int sorted = 0;
	for (int i = oddeven; i < size-oddeven; i += 2)
	{
		int minStep = arr[i] > arr[i + 1];
		int min = arr[i + minStep];
		int maxStep = arr[i] <= arr[i + 1];
		int max = arr[i + maxStep];

		arr[i] = min;
		arr[i + 1] = max;

		sorted += minStep - maxStep;
	}
	return sorted;
}

__host__
void sortCPU(int *arr, int size)
{
	int i = 0;
	int sorted = 1;
	while (sorted != (-size+1))
	{
		sorted = oddeven(arr, size, i % 2);
		sorted += oddeven(arr, size, (i+1) % 2);
		i += 2;
	}
}

//__device__
//int oddevenGPU(int *d_arr, int size, int oddeven, int blockSize, int startIndex, int endIndex)
//{
//	int sorted = 0;
//	for (int i = startIndex; i < endIndex; i += 2)
//	{
//		int minStep = d_arr[i] > d_arr[i + 1];
//		int min = d_arr[i + minStep];
//		int maxStep = d_arr[i] <= d_arr[i + 1];
//		int max = d_arr[i + maxStep];
//
//		d_arr[i] = min;
//		d_arr[i + 1] = max;
//
//		sorted += minStep - maxStep;
//	}
//	return sorted;
//}
//
//__global__
//void addKernel(int *d_arr, int *d_size, int *d_blockSize, int *d_sorted)
//{
//	int size = *d_size;
//	int blockSize = *d_blockSize;
//	int nrThreads = size / blockSize;
//	int elemInThread = size / nrThreads;
//	int shift = elemInThread % 2;
//
//	int i = 0;
//	int sorted = 0;
//	int oddeven = 0;
//	while (sorted != (-size + 1))
//	{
//		sorted = 0;
//
//
//		oddeven = i % 2;	//0 == odd, 1 == even
//		int startIndex = blockSize * threadIdx.x + oddeven + (shift * ((threadIdx.x + 1) % 2) * threadIdx.x != 0);
//		int endIndex = blockSize + blockSize * threadIdx.x - oddeven + shift * ((threadIdx.x + 1)%2);
//
//		sorted += oddevenGPU(d_arr, size, oddeven, blockSize, startIndex, endIndex);
//		__syncthreads();
//
//
//		oddeven = (i + 1) % 2;
//		startIndex = blockSize * threadIdx.x + oddeven;
//		endIndex = blockSize + blockSize * threadIdx.x - oddeven;
//
//		sorted += oddevenGPU(d_arr, size, oddeven, blockSize, startIndex, endIndex);
//		__syncthreads();
//		i += 2;
//	}
//}



//__global__
//void addKernel(int *d_arr, int *d_size, int *d_blocksize, int *d_sorted)
//{
//	int i = blockIdx.x * threadIdx.x + threadIdx.x * 2;
//
//	int swaps = 0;
//	int oddevenRun = 0;
//
//	do
//	{
//		swaps += (d_arr[i] > d_arr[i + 1]);
//
//		int minStep = d_arr[i] > d_arr[i + 1];
//		int min = d_arr[i + minStep];
//		int maxStep = d_arr[i] <= d_arr[i + 1];
//		int max = d_arr[i + maxStep];
//
//		d_arr[i] = min;
//		d_arr[i + 1] = max;
//
//		if (i % 2 == 0)
//			++i;
//		else
//			--i;
//
//		oddevenRun = (oddevenRun + 1) * (oddevenRun != 2);
//		swaps = swaps * (swaps != 2);
//
//		__syncthreads();
//
//		*d_sorted -= 1 * (swaps != 0 * oddevenRun != 2);
//
//	} while (*d_sorted > 0);
//}


//int main()
//{
//	srand((unsigned int)time(NULL));
//
//	int size = 100;
//	int *arr = (int*)malloc(size * sizeof(int));
//	int *d_arr, *d_size, *d_blockSize, *d_sorted;
//	createRandArr(arr, size, size*2);
//
//	int n = 2;
//	int blockSize = size / n;
//
//	printArr(arr, size);
//	
//	cudaMalloc(&d_arr, size * sizeof(int));
//	cudaMalloc(&d_size, sizeof(int));
//	cudaMalloc(&d_blockSize, sizeof(int));
//	cudaMalloc(&d_sorted, sizeof(int));
//
//	cudaMemcpy(d_arr, arr, size * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_size, &size, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_blockSize, &blockSize, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_sorted, &size, sizeof(int), cudaMemcpyHostToDevice);
//
//	int nr = size / blockSize;
//	
//	addKernel<<<1, (size/2)>>>(d_arr, d_size, d_blockSize, d_sorted);
//	cudaMemcpy(arr, d_arr, size * sizeof(int), cudaMemcpyDeviceToHost);
//
//	printArr(arr, size);
//
//	/*printArr(arr, size);
//	sortCPU(arr, size);
//	printArr(arr, size);*/
//
//	system("pause");
//
//	cudaFree(d_arr);
//	cudaFree(d_size);
//	free(arr);
//	return 0;
//}



__global__
void oddeven(int *arr, int flag)
{
	int d_flag = flag%2;
	int index = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	if ((index >= N - 1) && d_flag != 0) return;	//Out of bounds

	index += d_flag;

	int min = arr[index + (arr[index] > arr[index + 1])];
	int max = arr[index + (arr[index] <= arr[index + 1])];

	arr[index] = min;
	arr[index + 1] = max;
}

int main()
{
	int *arr;
	int *d_arr;
	int i;
	int size = sizeof(int) * N;
	srand((unsigned)time(NULL));

	arr = (int*)malloc(size);

	hipMalloc(&d_arr, size);

	createRandArr(arr, N, N * 2);

	//printArr(arr, N);

	double start_time = clock();

	hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

	for (i = 0; i<=N; i++)
	{
		oddeven<<<N / 2048, 1024>>>(d_arr, i);
	}

	hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);

	printf("\nExecution time: %lf seconds.\n", (clock() - start_time) / CLOCKS_PER_SEC);

	//printArr(arr, N);

	bool sorted = checkArr(arr, N);

	system("pause");

	return 0;
}