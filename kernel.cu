//#include "kernel.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"


void printArr(int *arr, int size)
{
	for (int i = 0; i < size - 1; ++i)
	{
		printf("%d, ", arr[i]);
	}
	printf("%d \n\n", arr[size - 1]);
}

void createRandArr(int *arr, int size, int maxVal)
{
	for (int i = 0; i < size; ++i)
	{
		int rnd = (rand() / (float)(RAND_MAX)) * maxVal;
		arr[i] = rnd;
	}
}

int oddeven(int *arr, int size, int oddeven)
{
	int sorted = 0;
	for (int i = oddeven; i < size-oddeven; i += 2)
	{
		int minStep = arr[i] > arr[i + 1];
		int min = arr[i + minStep];
		int maxStep = arr[i] <= arr[i + 1];
		int max = arr[i + maxStep];

		arr[i] = min;
		arr[i + 1] = max;

		sorted += minStep - maxStep;
	}
	return sorted;
}

void sortCPU(int *arr, int size)
{
	bool notSorted = true;
	int i = 0;
	int sorted = 1;
	while (sorted != (-size+1))
	{
		sorted = oddeven(arr, size, i % 2);
		sorted += oddeven(arr, size, (i+1) % 2);
		i += 2;
	}
}

__device__
int oddevenGPU(int *d_arr, int size, int oddeven, int blockSize)
{
	int sorted = 0;
	int start = threadIdx.x * blockSize + oddeven;
	int end = oddeven - blockSize + blockSize * threadIdx.x;
	for (int i = start; i < end; i += 2)
	{
		int minStep = d_arr[i] > d_arr[i + 1];
		int min = d_arr[i + minStep];
		int maxStep = d_arr[i] <= d_arr[i + 1];
		int max = d_arr[i + maxStep];

		d_arr[i] = min;
		d_arr[i + 1] = max;

		sorted += minStep - maxStep;
	}
	return sorted;
}

__global__
void addKernel(int *d_arr, int *d_size, int *d_blockSize)
{
	int size = *d_size;
	int blockSize = *d_blockSize;
	int nrThreads = size / blockSize;

	bool notSorted = true;
	int i = 0;
	int sorted = 1;
	while (sorted != (-size + 1))
	{
		sorted = 0;
		sorted += oddevenGPU(d_arr, size, i % 2, blockSize);
		__syncthreads();
		sorted += oddevenGPU(d_arr, size, (i + 1) % 2, blockSize);
		__syncthreads();
		i += 2;
	}
}

int main()
{
	srand((unsigned int)time(NULL));

	int size = 100;
	int setSize = 32;
	int *arr = (int*)malloc(size * sizeof(int));
	int *d_arr, *d_size, *d_blockSize;
	createRandArr(arr, size, size*2);

	printArr(arr, size);

	int set = size / 2;

	hipMalloc(&d_arr, size * sizeof(int));
	hipMalloc(&d_size, sizeof(int));
	hipMalloc(&d_blockSize, sizeof(int));

	hipMemcpy(d_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_blockSize, &set, sizeof(int), hipMemcpyHostToDevice);

	int nr = size / set;

	addKernel<<<1, 2>>>(d_arr, d_size, d_blockSize);
	hipMemcpy(arr, d_arr, size * sizeof(int), hipMemcpyDeviceToHost);

	printArr(arr, size);

	/*printArr(arr, size);
	sortCPU(arr, size);
	printArr(arr, size);*/

	system("pause");

	hipFree(d_arr);
	hipFree(d_size);
	free(arr);
	return 0;
}